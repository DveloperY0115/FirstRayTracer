#include "hip/hip_runtime.h"
//
// Created by dveloperY0115 on 1/8/2021.
//

#include "rtweekend-gpu.hpp"
#include "vector3.hpp"
#include "ray.hpp"
#include "sphere.hpp"
#include "hittable_list.hpp"
#include "camera.hpp"

#include "material.hpp"
#include "lambertian.hpp"
#include "metal.hpp"
#include "dielectric.hpp"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera, int nx, int ny) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vector3(0,0,-1), 0.5,
                               new lambertian(color(0.1, 0.2, 0.5)));
        d_list[1] = new sphere(vector3(0,-100.5,-1), 100,
                               new lambertian(color(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(vector3(1,0,-1), 0.5,
                               new metal(color(0.8, 0.6, 0.2), 0.0));
        d_list[3] = new sphere(vector3(-1,0,-1), 0.5,
                               new dielectric(1.5));
        d_list[4] = new sphere(vector3(-1,0,-1), -0.45,
                               new dielectric(1.5));
        *d_world  = new hittable_list(d_list,5);

        // initialize camera
        vector3 lookfrom(3, 3, 2);
        vector3 lookat(0, 0, -1);

        float dist_to_focus = (lookfrom - lookat).length();
        float aperture = 2.0f;
        *d_camera = new camera(
                lookfrom,
                lookat,
                vector3(0, 1, 0),
                20.0,
                float(nx)/float(ny),
                aperture,
                dist_to_focus
                );
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera **d_camera) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
    delete *d_camera;
}

__device__ vector3 ray_color(const ray& r, hittable **world, hiprandState *local_rand_state) {

    ray cur_ray = r;
    vector3 cur_attenuation = vector3(1.0, 1.0, 1.0);

    for (int i = 0; i < 50; i++) {
        hit_record rec;

        // while ray keeps bouncing off from objects, iterate
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vector3 attenuation;

            // if the material scatters the ray, track it
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            } else {
                // otherwise, it's a black body (an object which absorbs all lights)
                return vector3(0.0, 0.0, 0.0);
            }
        } else {
            // ray hit nothing. it's from the background, not an object in the scene
            vector3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vector3 color = (1.0f - t) * vector3(1.0, 1.0, 1.0) + t * vector3(0.5, 0.7, 1.0);
            return cur_attenuation * color;
        }
    }
    return vector3(0.0, 0.0, 0.0);  // exceeded recursion
}

//! \brief Initialize rendering system
//!
//! This function initializes random states allocated for each pixel on frame buffer
//! \param max_x width of the image
//! \param max_y height of the image
//! \param rand_state an array of hiprandState variables
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x >= max_x) || (y >= max_y)) return;

    unsigned int pixel_index = y * max_x + x;

    // each thread gets same seed, a different sequence number
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vector3* fb, int max_x, int max_y, int num_samples,
                       camera** cam, hittable **world, hiprandState* rand_state) {
    // get global pixel coordinate
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x >= max_x) || (y >= max_y)) return;   // don't render outside the image
    unsigned int pixel_index = y * max_x + x;
    hiprandState local_rand_state = rand_state[pixel_index];
    vector3 pixel_color = color(0, 0, 0);
    for (int sample_idx = 0; sample_idx < num_samples; sample_idx++) {
        float u = float(x + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(y + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        pixel_color += ray_color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    pixel_color /= float(num_samples);
    // gamma correction
    pixel_color[0] = sqrt(pixel_color[0]);
    pixel_color[1] = sqrt(pixel_color[1]);
    pixel_color[2] = sqrt(pixel_color[2]);

    fb[pixel_index] = pixel_color;
}

int main() {

    // Image
    const float aspect_ratio = 16.0 / 9.0;
    const int image_width = 1600;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    static const int num_samples = 50;

    // allocate Frame Buffer for rendering
    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vector3);
    vector3 *fb;
    checkCudaErrors(hipMallocManaged((void**) &fb, fb_size));

    // set random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**) &d_rand_state, num_pixels * sizeof(hiprandState)));

    // set world and camera
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    hittable **d_list;
    checkCudaErrors(hipMalloc((void**) &d_list, 5 * sizeof(hittable*)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **) &d_world, sizeof(hittable*)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // set dimensions of grid
    int tx = 8;
    int ty = 8;
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << num_samples << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    clock_t start, stop;
    start = clock();
    // render
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, image_width, image_height, num_samples, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;

    std::cerr << "took " << timer_seconds << " seconds.\n";

    // write output to a file
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            float r = fb[pixel_index].r();
            float g = fb[pixel_index].g();
            float b = fb[pixel_index].b();
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
    return 0;
}