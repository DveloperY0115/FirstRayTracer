#include "hip/hip_runtime.h"
//
// Created by dveloperY0115 on 1/8/2021.
//

#include <iostream>
#include "vector3.hpp"
#include "ray.hpp"

#define colorDim 3
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit_sphere(const point3& center, float radius, const ray& r) {
    vector3 oc = r.origin() - center;
    auto a = dot(r.direction(), r.direction());
    auto b = 2.0 * dot(oc, r.direction());
    auto c = dot(oc, oc) - radius*radius;
    auto discriminant = b*b - 4*a*c;
    return (discriminant > 0);
}

__device__ vector3 ray_color(const ray& r) {
    if (hit_sphere(point3(0, 0, -1), 0.5, r))
        return color(1, 0, 0);

    vector3 unit_direction = unit_vector(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0f);
    return (1.0f - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
}

__global__ void render(float* fb, int max_x, int max_y, vector3 lower_left_corner, vector3 horizontal,
                       vector3 vertical, vector3 origin) {
    // get global pixel coordinate
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x >= max_x) || (y >= max_y)) return;   // don't render outside the image
    unsigned int pixel_index = y * max_x * colorDim + x * colorDim;
    float u = float(x) / float(max_x);
    float v = float(y) / float(max_y);

    ray r = ray(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = ray_color(r)[0];
    fb[pixel_index + 1] = ray_color(r)[1];
    fb[pixel_index + 2] = ray_color(r)[2];
}

int main() {

    // Image
    const auto aspect_ratio = 16.0 / 9.0;
    const int image_width = 400;
    const int image_height = static_cast<int>(image_width / aspect_ratio);

    // Camera

    float viewport_height = 2.0;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_length = 1.0;

    auto origin = point3(0, 0, 0);
    auto horizontal = vector3(viewport_width, 0, 0);
    auto vertical = vector3(0, viewport_height, 0);
    auto lower_left_corner = origin - horizontal/2 - vertical/2 - vector3(0, 0, focal_length);

    int num_pixels = image_width * image_height;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate Frame Buffer for rendering
    float *fb;
    checkCudaErrors(hipMallocManaged((void**) &fb, fb_size));

    // set dimensions of grid
    int tx = 8;
    int ty = 8;

    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, image_width, image_height, lower_left_corner, horizontal, vertical, origin);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width * colorDim + i * colorDim;
            float r = fb[pixel_index];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));
    return 0;
}