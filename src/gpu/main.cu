#include "hip/hip_runtime.h"
//
// Created by dveloperY0115 on 1/8/2021.
//

#include <iostream>
#include "vector3.hpp"
#include "ray.hpp"
#include "sphere.hpp"
#include "hittable_list.hpp"

#define colorDim 3
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit_sphere(const point3& center, float radius, const ray& r) {
    vector3 oc = r.origin() - center;
    auto a = dot(r.direction(), r.direction());
    auto b = 2.0 * dot(oc, r.direction());
    auto c = dot(oc, oc) - radius*radius;
    auto discriminant = b*b - 4*a*c;
    return (discriminant > 0);
}

__global__ void create_world(hittable **d_list, hittable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vector3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vector3(0,-100.5,-1), 100);
        *d_world    = new hittable_list(d_list,2);
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
}

__device__ vector3 ray_color(const ray& r, hittable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f * vector3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }

    vector3 unit_direction = unit_vector(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0f);
    return (1.0f - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
}

__global__ void render(float* fb, int max_x, int max_y, vector3 lower_left_corner, vector3 horizontal,
                       vector3 vertical, vector3 origin, hittable **world) {
    // get global pixel coordinate
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x >= max_x) || (y >= max_y)) return;   // don't render outside the image
    unsigned int pixel_index = y * max_x * colorDim + x * colorDim;
    float u = float(x) / float(max_x);
    float v = float(y) / float(max_y);

    ray r = ray(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = ray_color(r, world)[0];
    fb[pixel_index + 1] = ray_color(r, world)[1];
    fb[pixel_index + 2] = ray_color(r, world)[2];
}

int main() {

    // Image
    const auto aspect_ratio = 16.0 / 9.0;
    const int image_width = 400;
    const int image_height = static_cast<int>(image_width / aspect_ratio);

    // Camera

    float viewport_height = 2.0;
    float viewport_width = aspect_ratio * viewport_height;
    float focal_length = 1.0;

    auto origin = point3(0, 0, 0);
    auto horizontal = vector3(viewport_width, 0, 0);
    auto vertical = vector3(0, viewport_height, 0);
    auto lower_left_corner = origin - horizontal/2 - vertical/2 - vector3(0, 0, focal_length);

    // world

    hittable **d_list;
        checkCudaErrors(hipMalloc((void**) &d_list, 2 * sizeof(hittable*)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **) &d_world, sizeof(hittable*)));
    create_world<<<1, 1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    int num_pixels = image_width * image_height;
    size_t fb_size = 3 * num_pixels * sizeof(float);

    // allocate Frame Buffer for rendering
    float *fb;
    checkCudaErrors(hipMallocManaged((void**) &fb, fb_size));

    // set dimensions of grid
    int tx = 8;
    int ty = 8;

    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, image_width, image_height, lower_left_corner, horizontal, vertical, origin, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width * colorDim + i * colorDim;
            float r = fb[pixel_index];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    free_world<<<1, 1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));
    return 0;
}